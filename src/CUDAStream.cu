#include "hip/hip_runtime.h"

#include "CUDAStream.h"

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

template <class T>
CUDAStream<T>::CUDAStream(const unsigned int ARRAY_SIZE)
{
  // Create device buffers
  hipMalloc(&d_a, ARRAY_SIZE*sizeof(T));
   
}

template <typename T>
__global__ void copy_kernel(const T * a, T * c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i];
}

template <class T>
void CUDAStream<T>::copy()
{
  copy_kernel<<<1024, 1024>>>(d_a, d_c);
}

template <class T>
void CUDAStream<T>::mul()
{
  return;
}

template <class T>
void CUDAStream<T>::add()
{
  return;
}

template <class T>
void CUDAStream<T>::triad()
{
  return;
}

template class CUDAStream<float>;
template class CUDAStream<double>;
