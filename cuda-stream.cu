
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cfloat>
#include <iomanip>
#include <cmath>

#include <hip/hip_runtime.h>

#define DATATYPE double
unsigned int ARRAY_SIZE = 50000000;
unsigned int NTIMES = 10;

size_t DATATYPE_SIZE = sizeof(double);
bool useFloat = false;

#define MIN(a,b) ((a) < (b)) ? (a) : (b)
#define MAX(a,b) ((a) > (b)) ? (a) : (b)

#define VERSION_STRING "0.0"

void parseArguments(int argc, char *argv[]);
std::string getDeviceName(int device);

struct invaliddevice : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Chosen device index is invalid";
    }
};

struct badntimes : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Chosen number of times is invalid, must be >= 2";
    }
};

// Code to check CUDA errors
void check_cuda_error(void)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr
            << "Error: "
            << hipGetErrorString(err)
            << std::endl;
            exit(err);
    }
}

void check_solution(void* a, void* b, void* c)
{
    // Generate correct solution
    double golda = 1.0;
    double goldb = 2.0;
    double goldc = 0.0;
    float goldaf = 1.0;
    float goldbf = 2.0;
    float goldcf = 0.0;

    const double scalar = 3.0;
    const float scalarf = 3.0;

    for (unsigned int i = 0; i < NTIMES; i++)
    {
        // Double
        goldc = golda;
        goldb = scalar * goldc;
        goldc = golda + goldb;
        golda = goldb + scalar * goldc;
        // Float
        goldcf = goldaf;
        goldbf = scalarf * goldcf;
        goldcf = goldaf + goldbf;
        goldaf = goldbf + scalarf * goldcf;
    }

    // Calculate average error
    double erra = 0.0;
    double errb = 0.0;
    double errc = 0.0;
    for (unsigned int i = 0; i < ARRAY_SIZE; i++)
    {
        if (useFloat)
        {
            erra += fabsf(((float*)a)[i] - goldaf);
            errb += fabsf(((float*)b)[i] - goldbf);
            errc += fabsf(((float*)c)[i] - goldcf);
        }
        else
        {
            erra += fabs(((double*)a)[i] - (double)golda);
            errb += fabs(((double*)b)[i] - (double)goldb);
            errc += fabs(((double*)c)[i] - (double)goldc);
        }
    }
    erra /= (double)ARRAY_SIZE;
    errb /= (double)ARRAY_SIZE;
    errc /= (double)ARRAY_SIZE;

    double epsi;
    if (useFloat) epsi = 1.0E-6;
    else epsi = 1.0E-13;

    if (erra > epsi)
        std::cout
            << "Validation failed on a[]. Average error " << erra
            << std::endl;
    if (errb > epsi)
        std::cout
            << "Validation failed on b[]. Average error " << errb
            << std::endl;
    if (errc > epsi)
        std::cout
            << "Validation failed on c[]. Average error " << errc
            << std::endl;
}


template <typename T>
__global__ void copy(const T * a, T * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i];
}

template <typename T>
__global__ void mul(T * b, const T * c)
{
    const T scalar = 3.0;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    b[i] = scalar * c[i];
}

template <typename T>
__global__ void add(const T * a, const T * b, T * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

template <typename T>
__global__ void triad(T * a, const T * b, const T * c)
{
    const T scalar = 3.0;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    a[i] = b[i] + scalar * c[i];
}

int deviceIndex = 0;

int main(int argc, char *argv[])
{

    // Print out run information
    std::cout
        << "GPU-STREAM" << std::endl
        << "Version: " << VERSION_STRING << std::endl
        << "Implementation: CUDA" << std::endl;

    try
    {
        parseArguments(argc, argv);

        if (NTIMES < 2) throw badntimes();

        std::cout << "Precision: ";
        if (useFloat) std::cout << "float";
        else std::cout << "double";
        std::cout << std::endl << std::endl;

        if (ARRAY_SIZE % 1024 != 0)
        {
            unsigned int OLD_ARRAY_SIZE = ARRAY_SIZE;
            ARRAY_SIZE -= ARRAY_SIZE % 1024;
            std::cout
                << "Warning: array size must divide 1024" << std::endl
                << "Resizing array from " << OLD_ARRAY_SIZE
                << " to " << ARRAY_SIZE << std::endl;
        }

        // Get precision (used to reset later)
        std::streamsize ss = std::cout.precision();

        // Display number of bytes in array
        std::cout << std::setprecision(1) << std::fixed
            << "Array size: " << ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0 << " MB"
            << " (=" << ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0/1024.0 << " GB)"
            << std::endl;
        std::cout << "Total size: " << 3*ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0 << " MB"
            << " (=" << 3*ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0/1024.0 << " GB)"
            << std::endl;

        // Reset precision
        std::cout.precision(ss);

        // Check device index is in range
        int count;
        hipGetDeviceCount(&count);
        check_cuda_error();
        if (deviceIndex >= count) throw invaliddevice();
        hipSetDevice(deviceIndex);
        check_cuda_error();

        // Print out device name
        std::cout << "Using CUDA device " << getDeviceName(deviceIndex) << std::endl;


        // Create host vectors
        void * h_a = malloc(ARRAY_SIZE*DATATYPE_SIZE);
        void * h_b = malloc(ARRAY_SIZE*DATATYPE_SIZE);
        void * h_c = malloc(ARRAY_SIZE*DATATYPE_SIZE);

        // Initilise arrays
        for (unsigned int i = 0; i < ARRAY_SIZE; i++)
        {
            if (useFloat)
            {
                ((float*)h_a)[i] = 1.0;
                ((float*)h_b)[i] = 2.0;
                ((float*)h_c)[i] = 0.0;
            }
            else
            {
                ((double*)h_a)[i] = 1.0;
                ((double*)h_b)[i] = 2.0;
                ((double*)h_c)[i] = 0.0;
            }
        }

        // Create device buffers
        void * d_a, * d_b, *d_c;
        hipMalloc(&d_a, ARRAY_SIZE*DATATYPE_SIZE);
        check_cuda_error();
        hipMalloc(&d_b, ARRAY_SIZE*DATATYPE_SIZE);
        check_cuda_error();
        hipMalloc(&d_c, ARRAY_SIZE*DATATYPE_SIZE);
        check_cuda_error();

        // Copy host memory to device
        hipMemcpy(d_a, h_a, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
        check_cuda_error();
        hipMemcpy(d_b, h_b, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
        check_cuda_error();
        hipMemcpy(d_c, h_c, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
        check_cuda_error();

        // Make sure the copies are finished
        hipDeviceSynchronize();
        check_cuda_error();

        // List of times
        std::vector< std::vector<double> > timings;

        // Declare timers
        std::chrono::high_resolution_clock::time_point t1, t2;

        // Main loop
        for (unsigned int k = 0; k < NTIMES; k++)
        {
            std::vector<double> times;
            t1 = std::chrono::high_resolution_clock::now();
            if (useFloat)
                copy<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_c);
            else
                copy<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_c);
            check_cuda_error();
            hipDeviceSynchronize();
            check_cuda_error();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            if (useFloat)
                mul<<<ARRAY_SIZE/1024, 1024>>>((float*)d_b, (float*)d_c);
            else
                mul<<<ARRAY_SIZE/1024, 1024>>>((double*)d_b, (double*)d_c);
            check_cuda_error();
            hipDeviceSynchronize();
            check_cuda_error();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            if (useFloat)
                add<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_b, (float*)d_c);
            else
                add<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_b, (double*)d_c);
            check_cuda_error();
            hipDeviceSynchronize();
            check_cuda_error();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            if (useFloat)
                triad<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_b, (float*)d_c);
            else
                triad<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_b, (double*)d_c);
            check_cuda_error();
            hipDeviceSynchronize();
            check_cuda_error();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

            timings.push_back(times);

        }

        // Check solutions
        hipMemcpy(h_a, d_a, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
        check_cuda_error();
        hipMemcpy(h_b, d_b, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
        check_cuda_error();
        hipMemcpy(h_c, d_c, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
        check_cuda_error();
        check_solution(h_a, h_b, h_c);

        // Crunch results
        size_t sizes[4] = {
            2 * DATATYPE_SIZE * ARRAY_SIZE,
            2 * DATATYPE_SIZE * ARRAY_SIZE,
            3 * DATATYPE_SIZE * ARRAY_SIZE,
            3 * DATATYPE_SIZE * ARRAY_SIZE
        };
        double min[4] = {DBL_MAX, DBL_MAX, DBL_MAX, DBL_MAX};
        double max[4] = {0.0, 0.0, 0.0, 0.0};
        double avg[4] = {0.0, 0.0, 0.0, 0.0};
        // Ignore first result
        for (unsigned int i = 1; i < NTIMES; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                avg[j] += timings[i][j];
                min[j] = MIN(min[j], timings[i][j]);
                max[j] = MAX(max[j], timings[i][j]);
            }
        }
        for (int j = 0; j < 4; j++)
            avg[j] /= (double)(NTIMES-1);

        // Display results
        std::string labels[] = {"Copy", "Mul", "Add", "Triad"};
        std::cout
            << std::left << std::setw(12) << "Function"
            << std::left << std::setw(12) << "MBytes/sec"
            << std::left << std::setw(12) << "Min (sec)"
            << std::left << std::setw(12) << "Max"
            << std::left << std::setw(12) << "Average"
            << std::endl;
        for (int j = 0; j < 4; j++)
        {
            std::cout
                << std::left << std::setw(12) << labels[j]
                << std::left << std::setw(12) << std::setprecision(3) << 1.0E-06 * sizes[j]/min[j]
                << std::left << std::setw(12) << std::setprecision(5) << min[j]
                << std::left << std::setw(12) << std::setprecision(5) << max[j]
                << std::left << std::setw(12) << std::setprecision(5) << avg[j]
                << std::endl;
        }

    }
    catch (std::exception& e)
    {
        std::cerr
            << "Error: "
            << e.what()
            << std::endl;
    }
}

std::string getDeviceName(int device)
{
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    check_cuda_error();
    return std::string(prop.name);
}


int parseUInt(const char *str, unsigned int *output)
{
    char *next;
    *output = strtoul(str, &next, 10);
    return !strlen(next);
}

int parseInt(const char *str, int *output)
{
    char *next;
    *output = strtol(str, &next, 10);
    return !strlen(next);
}

void parseArguments(int argc, char *argv[])
{
    for (int i = 1; i < argc; i++)
    {
        if (!strcmp(argv[i], "--list"))
        {
            // Get number of devices
            int count;
            hipGetDeviceCount(&count);
            check_cuda_error();

            // Print device names
            if (count == 0)
            {
                std::cout << "No devices found." << std::endl;
            }
            else
            {
                std::cout << std::endl;
                std::cout << "Devices:" << std::endl;
                for (int i = 0; i < count; i++)
                {
                    std::cout << i << ": " << getDeviceName(i) << std::endl;
                    check_cuda_error();
                }
                std::cout << std::endl;
            }
            exit(0);
        }
        else if (!strcmp(argv[i], "--device"))
        {
            if (++i >= argc || !parseInt(argv[i], &deviceIndex))
            {
                std::cout << "Invalid device index" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--arraysize") || !strcmp(argv[i], "-s"))
        {
            if (++i >= argc || !parseUInt(argv[i], &ARRAY_SIZE))
            {
                std::cout << "Invalid array size" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--numtimes") || !strcmp(argv[i], "-n"))
        {
            if (++i >= argc || !parseUInt(argv[i], &NTIMES))
            {
                std::cout << "Invalid number of times" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--float"))
        {
            useFloat = true;
            DATATYPE_SIZE = sizeof(float);
        }
        else if (!strcmp(argv[i], "--help") || !strcmp(argv[i], "-h"))
        {
            std::cout << std::endl;
            std::cout << "Usage: ./gpu-stream-cuda [OPTIONS]" << std::endl << std::endl;
            std::cout << "Options:" << std::endl;
            std::cout << "  -h  --help               Print the message" << std::endl;
            std::cout << "      --list               List available devices" << std::endl;
            std::cout << "      --device     INDEX   Select device at INDEX" << std::endl;
            std::cout << "  -s  --arraysize  SIZE    Use SIZE elements in the array" << std::endl;
            std::cout << "  -n  --numtimes   NUM     Run the test NUM times (NUM >= 2)" << std::endl;
            std::cout << "      --float              Use floats (rather than doubles)" << std::endl;
            std::cout << std::endl;
            exit(0);
        }
        else
        {
            std::cout << "Unrecognized argument '" << argv[i] << "' (try '--help')"
                << std::endl;
            exit(1);
        }
    }
}
