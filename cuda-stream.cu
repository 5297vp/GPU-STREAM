
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cfloat>
#include <iomanip>
#include <cmath>

#include <hip/hip_runtime.h>

#define DATATYPE double
unsigned int ARRAY_SIZE = 50000000;
unsigned int NTIMES = 10;

#define MIN(a,b) ((a) < (b)) ? (a) : (b)
#define MAX(a,b) ((a) > (b)) ? (a) : (b)

#define VERSION_STRING "0.0"

void parseArguments(int argc, char *argv[]);
std::string getDeviceName();

struct badtype : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Datatype is not 4 or 8";
    }
};

struct invaliddevice : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Chosen device index is invalid";
    }
};

struct badntimes : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Chosen number of times is invalid, must be >= 2";
    }
};

size_t sizes[4] = {
    2 * sizeof(DATATYPE) * ARRAY_SIZE,
    2 * sizeof(DATATYPE) * ARRAY_SIZE,
    3 * sizeof(DATATYPE) * ARRAY_SIZE,
    3 * sizeof(DATATYPE) * ARRAY_SIZE
};

void check_solution(DATATYPE * a, DATATYPE * b, DATATYPE * c)
{
    // Generate correct solution
    DATATYPE golda = 1.0;
    DATATYPE goldb = 2.0;
    DATATYPE goldc = 0.0;

    const DATATYPE scalar = 3.0;

    for (unsigned int i = 0; i < NTIMES; i++)
    {
        goldc = golda;
        goldb = scalar * goldc;
        goldc = golda + goldb;
        golda = goldb + scalar * goldc;
    }

    // Calculate average error
    double erra = 0.0;
    double errb = 0.0;
    double errc = 0.0;
    for (unsigned int i = 0; i < ARRAY_SIZE; i++)
    {
        erra += fabs(a[i] - golda);
        errb += fabs(b[i] - goldb);
        errc += fabs(c[i] - goldc);
    }
    erra /= (double)ARRAY_SIZE;
    errb /= (double)ARRAY_SIZE;
    errc /= (double)ARRAY_SIZE;

    double epsi;
    if (sizeof(DATATYPE) == 4) epsi = 1.0E-6;
    else if (sizeof(DATATYPE) == 8) epsi = 1.0E-13;
    else throw badtype();

    if (erra > epsi)
        std::cout
            << "Validation failed on a[]. Average error " << erra
            << std::endl;
    if (errb > epsi)
        std::cout
            << "Validation failed on b[]. Average error " << errb
            << std::endl;
    if (errc > epsi)
        std::cout
            << "Validation failed on c[]. Average error " << errc
            << std::endl;
}

const DATATYPE scalar = 3.0;

__global__ void copy(const DATATYPE * a, DATATYPE * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i];
}

__global__ void mul(DATATYPE * b, const DATATYPE * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    b[i] = scalar * c[i];
}

__global__ void add(const DATATYPE * a, const DATATYPE * b, DATATYPE * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void triad(DATATYPE * a, const DATATYPE * b, const DATATYPE * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    a[i] = b[i] + scalar * c[i];
}

int deviceIndex = 0;

int main(int argc, char *argv[])
{

    // Print out run information
    std::cout
        << "GPU-STREAM" << std::endl
        << "Version: " << VERSION_STRING << std::endl
        << "Implementation: CUDA" << std::endl << std::endl;

    if (ARRAY_SIZE % 1024 != 0)
    {
        unsigned int OLD_ARRAY_SIZE = ARRAY_SIZE;
        ARRAY_SIZE -= ARRAY_SIZE % 1024;
        std::cout
            << "Warning: array size must divide 1024" << std::endl
            << "Resizing array from " << OLD_ARRAY_SIZE
            << " to " << ARRAY_SIZE << std::endl;
    }

    try
    {
        parseArguments(argc, argv);

        if (NTIMES < 2) throw badntimes();

        // Check device index is in range
        int count;
        hipGetDeviceCount(&count);
        if (deviceIndex >= count) throw invaliddevice();
        hipSetDevice(deviceIndex);

        // Print out device name
        std::cout << "Using CUDA device " << getDeviceName() << std::endl;


        // Create host vectors
        DATATYPE * h_a = (DATATYPE *) malloc(ARRAY_SIZE*sizeof(DATATYPE));
        DATATYPE * h_b = (DATATYPE *) malloc(ARRAY_SIZE*sizeof(DATATYPE));
        DATATYPE * h_c = (DATATYPE *) malloc(ARRAY_SIZE*sizeof(DATATYPE));

        // Initilise host vectors
        for (unsigned int i = 0; i < ARRAY_SIZE; i++)
        {
            h_a[i] = 1.0;
            h_b[i] = 2.0;
            h_c[i] = 0.0;
        }

        // Create device buffers
        DATATYPE * d_a, * d_b, *d_c;
        hipMalloc(&d_a, ARRAY_SIZE*sizeof(DATATYPE));
        hipMalloc(&d_b, ARRAY_SIZE*sizeof(DATATYPE));
        hipMalloc(&d_c, ARRAY_SIZE*sizeof(DATATYPE));

        // Copy host memory to device
        hipMemcpy(d_a, h_a, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyHostToDevice);
        hipMemcpy(d_c, h_c, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyHostToDevice);

        // Make sure the copies are finished
        hipDeviceSynchronize();

        // List of times
        std::vector< std::vector<double> > timings;

        // Declare timers
        std::chrono::high_resolution_clock::time_point t1, t2;

        // Main loop
        for (unsigned int k = 0; k < NTIMES; k++)
        {
            std::vector<double> times;
            t1 = std::chrono::high_resolution_clock::now();
            copy<<<ARRAY_SIZE/1024, 1024>>>(d_a, d_c);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            mul<<<ARRAY_SIZE/1024, 1024>>>(d_b, d_c);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            add<<<ARRAY_SIZE/1024, 1024>>>(d_a, d_b, d_c);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            triad<<<ARRAY_SIZE/1024, 1024>>>(d_a, d_b, d_c);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

            timings.push_back(times);

        }

        // Check solutions
        hipMemcpy(h_a, d_a, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyDeviceToHost);
        hipMemcpy(h_b, d_b, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyDeviceToHost);
        hipMemcpy(h_c, d_c, ARRAY_SIZE*sizeof(DATATYPE), hipMemcpyDeviceToHost);
        check_solution(h_a, h_b, h_c);

        // Crunch results
        double min[4] = {DBL_MAX, DBL_MAX, DBL_MAX, DBL_MAX};
        double max[4] = {0.0, 0.0, 0.0, 0.0};
        double avg[4] = {0.0, 0.0, 0.0, 0.0};
        // Ignore first result
        for (unsigned int i = 1; i < NTIMES; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                avg[j] += timings[i][j];
                min[j] = MIN(min[j], timings[i][j]);
                max[j] = MAX(max[j], timings[i][j]);
            }
        }
        for (int j = 0; j < 4; j++)
            avg[j] /= (double)(NTIMES-1);

        // Display results
        std::string labels[] = {"Copy", "Mul", "Add", "Triad"};
        std::cout
            << std::left << std::setw(12) << "Function"
            << std::left << std::setw(12) << "MBytes/sec"
            << std::left << std::setw(12) << "Min (sec)"
            << std::left << std::setw(12) << "Max"
            << std::left << std::setw(12) << "Average"
            << std::endl;
        for (int j = 0; j < 4; j++)
        {
            std::cout
                << std::left << std::setw(12) << labels[j]
                << std::left << std::setw(12) << 1.0E-06 * sizes[j]/min[j]
                << std::left << std::setw(12) << min[j]
                << std::left << std::setw(12) << max[j]
                << std::left << std::setw(12) << avg[j]
                << std::endl;
        }

    }
    catch (std::exception& e)
    {
        std::cerr
            << "Error: "
            << e.what()
            << std::endl;
    }
}

std::string getDeviceName()
{
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    return std::string(prop.name);
}


int parseUInt(const char *str, unsigned int *output)
{
    char *next;
    *output = strtoul(str, &next, 10);
    return !strlen(next);
}

int parseInt(const char *str, int *output)
{
    char *next;
    *output = strtol(str, &next, 10);
    return !strlen(next);
}

void parseArguments(int argc, char *argv[])
{
    for (int i = 1; i < argc; i++)
    {
        if (!strcmp(argv[i], "--list"))
        {
            // Get number of devices
            int count;
            hipGetDeviceCount(&count);

            // Print device names
            if (count == 0)
            {
                std::cout << "No devices found." << std::endl;
            }
            else
            {
                std::cout << std::endl;
                std::cout << "Devices:" << std::endl;
                for (int i = 0; i < count; i++)
                {
                    hipSetDevice(i);
                    std::cout << i << ": " << getDeviceName() << std::endl;
                }
                std::cout << std::endl;
            }
            exit(0);
        }
        else if (!strcmp(argv[i], "--device"))
        {
            if (++i >= argc || !parseInt(argv[i], &deviceIndex))
            {
                std::cout << "Invalid device index" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--arraysize") || !strcmp(argv[i], "-s"))
        {
            if (++i >= argc || !parseUInt(argv[i], &ARRAY_SIZE))
            {
                std::cout << "Invalid array size" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--numtimes") || !strcmp(argv[i], "-n"))
        {
            if (++i >= argc || !parseUInt(argv[i], &NTIMES))
            {
                std::cout << "Invalid number of times" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--help") || !strcmp(argv[i], "-h"))
        {
            std::cout << std::endl;
            std::cout << "Usage: ./gpu-stream-ocl [OPTIONS]" << std::endl << std::endl;
            std::cout << "Options:" << std::endl;
            std::cout << "  -h  --help               Print the message" << std::endl;
            std::cout << "      --list               List available devices" << std::endl;
            std::cout << "      --device     INDEX   Select device at INDEX" << std::endl;
            std::cout << "  -s  --arraysize  SIZE    Use SIZE elements in the array" << std::endl;
            std::cout << "  -n  --numtimes   NUM     Run the test NUM times (NUM >= 2)" << std::endl;
            std::cout << std::endl;
            exit(0);
        }
        else
        {
            std::cout << "Unrecognized argument '" << argv[i] << "' (try '--help')"
                << std::endl;
            exit(1);
        }
    }
}
