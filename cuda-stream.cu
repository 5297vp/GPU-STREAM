#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cfloat>
#include <iomanip>
#include <cmath>

#include <hip/hip_runtime.h>

#define DATATYPE double
unsigned int ARRAY_SIZE = 50000000;
#define NTIMES 10

#define MIN(a,b) ((a) < (b)) ? (a) : (b)
#define MAX(a,b) ((a) > (b)) ? (a) : (b)

#define VERSION_STRING "0.0"

void parseArguments(int argc, char *argv[]);
std::string getDeviceName();

struct badtype : public std::exception
{
    virtual const char * what () const throw ()
    {
        return "Datatype is not 4 or 8";
    }
};


size_t sizes[4] = {
    2 * sizeof(DATATYPE) * ARRAY_SIZE,
    2 * sizeof(DATATYPE) * ARRAY_SIZE,
    3 * sizeof(DATATYPE) * ARRAY_SIZE,
    3 * sizeof(DATATYPE) * ARRAY_SIZE
};

void check_solution(std::vector<DATATYPE>& a, std::vector<DATATYPE>& b, std::vector<DATATYPE>& c)
{
    // Generate correct solution
    DATATYPE golda = 1.0;
    DATATYPE goldb = 2.0;
    DATATYPE goldc = 0.0;

    const DATATYPE scalar = 3.0;

    for (unsigned int i = 0; i < NTIMES; i++)
    {
        goldc = golda;
        goldb = scalar * goldc;
        goldc = golda + goldb;
        golda = goldb + scalar * goldc;
    }

    // Calculate average error
    double erra = 0.0;
    double errb = 0.0;
    double errc = 0.0;
    for (unsigned int i = 0; i < ARRAY_SIZE; i++)
    {
        erra += fabs(a[i] - golda);
        errb += fabs(b[i] - goldb);
        errc += fabs(c[i] - goldc);
    }
    erra /= (double)ARRAY_SIZE;
    errb /= (double)ARRAY_SIZE;
    errc /= (double)ARRAY_SIZE;

    double epsi;
    if (sizeof(DATATYPE) == 4) epsi = 1.0E-6;
    else if (sizeof(DATATYPE) == 8) epsi = 1.0E-13;
    else throw badtype();

    if (erra > epsi)
        std::cout
            << "Validation failed on a[]. Average error " << erra
            << std::endl;
    if (errb > epsi)
        std::cout
            << "Validation failed on b[]. Average error " << errb
            << std::endl;
    if (errc > epsi)
        std::cout
            << "Validation failed on c[]. Average error " << errc
            << std::endl;
}

const DATATYPE scalar = 3.0;

// __global__ void copy(const DATATYPE * restrict a, DATATYPE * restrict c)
// {
//     const int i = blockDim.x * blockIdx.x + threadIdx.x;
//     c[i] = a[i];
// }

// __global__ void mul(DATATYPE * restrict b, const DATATYPE * restrict c)
// {
//     const int i = blockDim.x * blockIdx.x + threadIdx.x;
//     b[i] = scalar * c[i];
// }

// __global__ void add(const DATATYPE * restrict a, const DATATYPE * restrict b, DATATYPE * restrict c)
// {
//     const int i = blockDim.x * blockIdx.x + threadIdx.x;
//     c[i] = a[i] + b[i];
// }

// __global__ void triad(DATATYPE * restrict a, const DATATYPE * restrict b, const DATATYPE * restrict c)
// {
//     const int i = blockDim.x * blockIdx.x + threadIdx.x;
//     a[i] = b[i] + scalar * c[i];
// }

cl_uint deviceIndex = 0;

int main(int argc, char *argv[])
{

    // Print out run information
    std::cout
        << "GPU-STREAM" << std::endl
        << "Version: " << VERSION_STRING << std::endl
        << "Implementation: OpenCL" << std::endl << std::endl;

    try
    {
        parseArguments(argc, argv);

        // Print out device name
        std::cout << "Using CUDA device " << getDeviceName() << std::endl;


        // Create host vectors
        std::vector<DATATYPE> h_a(ARRAY_SIZE, 1.0);
        std::vector<DATATYPE> h_b(ARRAY_SIZE, 2.0);
        std::vector<DATATYPE> h_c(ARRAY_SIZE, 0.0);

        // Create device buffers


        // Copy host memory to device


        // Make sure the copies are finished


        // List of times
        std::vector< std::vector<double> > timings;

        // Declare timers
        std::chrono::high_resolution_clock::time_point t1, t2;

        // Main loop
        for (unsigned int k = 0; k < NTIMES; k++)
        {
            /*std::vector<double> times;
            t1 = std::chrono::high_resolution_clock::now();
            copy(
                cl::EnqueueArgs(
                queue,
                cl::NDRange(ARRAY_SIZE)),
                d_a, d_c);
            queue.finish();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            mul(
                cl::EnqueueArgs(
                queue,
                cl::NDRange(ARRAY_SIZE)),
                d_b, d_c);
            queue.finish();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            add(
                cl::EnqueueArgs(
                queue,
                cl::NDRange(ARRAY_SIZE)),
                d_a, d_b, d_c);
            queue.finish();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


            t1 = std::chrono::high_resolution_clock::now();
            triad(
                cl::EnqueueArgs(
                queue,
                cl::NDRange(ARRAY_SIZE)),
                d_a, d_b, d_c);
            queue.finish();
            t2 = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

            timings.push_back(times);*/

        }

        // Check solutions

        check_solution(h_a, h_b, h_c);

        // Crunch results
        double min[4] = {DBL_MAX, DBL_MAX, DBL_MAX, DBL_MAX};
        double max[4] = {0.0, 0.0, 0.0, 0.0};
        double avg[4] = {0.0, 0.0, 0.0, 0.0};
        // Ignore first result
        for (unsigned int i = 1; i < NTIMES; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                avg[j] += timings[i][j];
                min[j] = MIN(min[j], timings[i][j]);
                max[j] = MAX(max[j], timings[i][j]);
            }
        }
        for (int j = 0; j < 4; j++)
            avg[j] /= (double)(NTIMES-1);

        // Display results
        std::string labels[] = {"Copy", "Mul", "Add", "Triad"};
        std::cout
            << std::left << std::setw(12) << "Function"
            << std::left << std::setw(12) << "MBytes/sec"
            << std::left << std::setw(12) << "Min (sec)"
            << std::left << std::setw(12) << "Max"
            << std::left << std::setw(12) << "Average"
            << std::endl;
        for (int j = 0; j < 4; j++)
        {
            std::cout
                << std::left << std::setw(12) << labels[j]
                << std::left << std::setw(12) << 1.0E-06 * sizes[j]/min[j]
                << std::left << std::setw(12) << min[j]
                << std::left << std::setw(12) << max[j]
                << std::left << std::setw(12) << avg[j]
                << std::endl;
        }

    }
    catch (std::exception& e)
    {
        std::cerr
            << "Error: "
            << e.what()
            << std::endl;
    }
}

unsigned getDeviceList()
{

  // // Enumerate devices
  // for (unsigned int i = 0; i < platforms.size(); i++)
  // {
  //   std::vector<cl::Device> plat_devices;
  //   platforms[i].getDevices(CL_DEVICE_TYPE_ALL, &plat_devices);
  //   devices.insert(devices.end(), plat_devices.begin(), plat_devices.end());
  // }

  // return devices.size();
    return 0;
}

std::string getDeviceName()
{
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    return std::string(prop.name);
}


int parseUInt(const char *str, cl_uint *output)
{
    char *next;
    *output = strtoul(str, &next, 10);
    return !strlen(next);
}

void parseArguments(int argc, char *argv[])
{
    for (int i = 1; i < argc; i++)
    {
        if (!strcmp(argv[i], "--list"))
        {
            // Get list of devices
            /*std::vector<cl::Device> devices;
            getDeviceList(devices);

            // Print device names
            if (devices.size() == 0)
            {
                std::cout << "No devices found." << std::endl;
            }
            else
            {
                std::cout << std::endl;
                std::cout << "Devices:" << std::endl;
                for (unsigned i = 0; i < devices.size(); i++)
                {
                    std::cout << i << ": " << getDeviceName(devices[i]) << std::endl;
                }
                std::cout << std::endl;
            }*/
            exit(0);
        }
        else if (!strcmp(argv[i], "--device"))
        {
            if (++i >= argc || !parseUInt(argv[i], &deviceIndex))
            {
                std::cout << "Invalid device index" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--arraysize") || !strcmp(argv[i], "-s"))
        {
            if (++i >= argc || !parseUInt(argv[i], &ARRAY_SIZE))
            {
                std::cout << "Invalid array size" << std::endl;
                exit(1);
            }
        }
        else if (!strcmp(argv[i], "--help") || !strcmp(argv[i], "-h"))
        {
            std::cout << std::endl;
            std::cout << "Usage: ./gpu-stream-ocl [OPTIONS]" << std::endl << std::endl;
            std::cout << "Options:" << std::endl;
            std::cout << "  -h  --help               Print the message" << std::endl;
            std::cout << "      --list               List available devices" << std::endl;
            std::cout << "      --device     INDEX   Select device at INDEX" << std::endl;
            std::cout << "  -s  --arraysize  SIZE    Use SIZE elements in the array" << std::endl;
            std::cout << std::endl;
            exit(0);
        }
        else
        {
            std::cout << "Unrecognized argument '" << argv[i] << "' (try '--help')"
                << std::endl;
            exit(1);
        }
    }
}
