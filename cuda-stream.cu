#include "hip/hip_runtime.h"
/*=============================================================================
*------------------------------------------------------------------------------
* Copyright 2015: Tom Deakin, Simon McIntosh-Smith, University of Bristol HPC
* Based on John D. McCalpin’s original STREAM benchmark for CPUs
*------------------------------------------------------------------------------
* License:
*  1. You are free to use this program and/or to redistribute
*     this program.
*  2. You are free to modify this program for your own use,
*     including commercial use, subject to the publication
*     restrictions in item 3.
*  3. You are free to publish results obtained from running this
*     program, or from works that you derive from this program,
*     with the following limitations:
*     3a. In order to be referred to as "GPU-STREAM benchmark results",
*         published results must be in conformance to the GPU-STREAM
*         Run Rules published at
*         http://github.com/UoB-HPC/GPU-STREAM/wiki/Run-Rules
*         and incorporated herein by reference.
*         The copyright holders retain the
*         right to determine conformity with the Run Rules.
*     3b. Results based on modified source code or on runs not in
*         accordance with the GPU-STREAM Run Rules must be clearly
*         labelled whenever they are published.  Examples of
*         proper labelling include:
*         "tuned GPU-STREAM benchmark results" 
*         "based on a variant of the GPU-STREAM benchmark code"
*         Other comparable, clear and reasonable labelling is
*         acceptable.
*     3c. Submission of results to the GPU-STREAM benchmark web site
*         is encouraged, but not required.
*  4. Use of this program or creation of derived works based on this
*     program constitutes acceptance of these licensing restrictions.
*  5. Absolutely no warranty is expressed or implied.
*———————————————————————————————————-----------------------------------------*/


#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cfloat>
#include <cmath>

#include <hip/hip_runtime.h>
#include "common.h"

std::string getDeviceName(int device);

// Code to check CUDA errors
void check_cuda_error(void)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr
            << "Error: "
            << hipGetErrorString(err)
            << std::endl;
            exit(err);
    }
}

template <typename T>
__global__ void copy(const T * a, T * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i];
}

template <typename T>
__global__ void mul(T * b, const T * c)
{
    const T scalar = 3.0;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    b[i] = scalar * c[i];
}

template <typename T>
__global__ void add(const T * a, const T * b, T * c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

template <typename T>
__global__ void triad(T * a, const T * b, const T * c)
{
    const T scalar = 3.0;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    a[i] = b[i] + scalar * c[i];
}

int main(int argc, char *argv[])
{

    // Print out run information
    std::cout
        << "GPU-STREAM" << std::endl
        << "Version: " << VERSION_STRING << std::endl
        << "Implementation: CUDA" << std::endl;

    parseArguments(argc, argv);

    if (NTIMES < 2) throw badntimes();

    std::cout << "Precision: ";
    if (useFloat) std::cout << "float";
    else std::cout << "double";
    std::cout << std::endl << std::endl;

    std::cout << "Running kernels " << NTIMES << " times" << std::endl;

    if (ARRAY_SIZE % 1024 != 0)
    {
        unsigned int OLD_ARRAY_SIZE = ARRAY_SIZE;
        ARRAY_SIZE -= ARRAY_SIZE % 1024;
        std::cout
            << "Warning: array size must divide 1024" << std::endl
            << "Resizing array from " << OLD_ARRAY_SIZE
            << " to " << ARRAY_SIZE << std::endl;
        if (ARRAY_SIZE == 0) throw badarraysize();
    }

    // Get precision (used to reset later)
    std::streamsize ss = std::cout.precision();

    size_t DATATYPE_SIZE;

    if (useFloat)
    {
        DATATYPE_SIZE = sizeof(float);
    }
    else
    {
        DATATYPE_SIZE = sizeof(double);
    }

    // Display number of bytes in array
    std::cout << std::setprecision(1) << std::fixed
        << "Array size: " << ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0 << " MB"
        << " (=" << ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0/1024.0 << " GB)"
        << std::endl;
    std::cout << "Total size: " << 3.0*ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0 << " MB"
        << " (=" << 3.0*ARRAY_SIZE*DATATYPE_SIZE/1024.0/1024.0/1024.0 << " GB)"
        << std::endl;

    // Reset precision
    std::cout.precision(ss);

    // Check device index is in range
    int count;
    hipGetDeviceCount(&count);
    check_cuda_error();
    if (deviceIndex >= count) throw invaliddevice();
    hipSetDevice(deviceIndex);
    check_cuda_error();

    // Print out device name
    std::cout << "Using CUDA device " << getDeviceName(deviceIndex) << std::endl;


    // Create host vectors
    void * h_a = malloc(ARRAY_SIZE*DATATYPE_SIZE);
    void * h_b = malloc(ARRAY_SIZE*DATATYPE_SIZE);
    void * h_c = malloc(ARRAY_SIZE*DATATYPE_SIZE);

    // Initilise arrays
    for (unsigned int i = 0; i < ARRAY_SIZE; i++)
    {
        if (useFloat)
        {
            ((float*)h_a)[i] = 1.0f;
            ((float*)h_b)[i] = 2.0f;
            ((float*)h_c)[i] = 0.0f;
        }
        else
        {
            ((double*)h_a)[i] = 1.0;
            ((double*)h_b)[i] = 2.0;
            ((double*)h_c)[i] = 0.0;
        }
    }

    // Create device buffers
    void * d_a, * d_b, *d_c;
    hipMalloc(&d_a, ARRAY_SIZE*DATATYPE_SIZE);
    check_cuda_error();
    hipMalloc(&d_b, ARRAY_SIZE*DATATYPE_SIZE);
    check_cuda_error();
    hipMalloc(&d_c, ARRAY_SIZE*DATATYPE_SIZE);
    check_cuda_error();

    // Copy host memory to device
    hipMemcpy(d_a, h_a, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
    check_cuda_error();
    hipMemcpy(d_b, h_b, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
    check_cuda_error();
    hipMemcpy(d_c, h_c, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyHostToDevice);
    check_cuda_error();

    // Make sure the copies are finished
    hipDeviceSynchronize();
    check_cuda_error();

    // List of times
    std::vector< std::vector<double> > timings;

    // Declare timers
    std::chrono::high_resolution_clock::time_point t1, t2;

    // Main loop
    for (unsigned int k = 0; k < NTIMES; k++)
    {
        std::vector<double> times;
        t1 = std::chrono::high_resolution_clock::now();
        if (useFloat)
            copy<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_c);
        else
            copy<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_c);
        check_cuda_error();
        hipDeviceSynchronize();
        check_cuda_error();
        t2 = std::chrono::high_resolution_clock::now();
        times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


        t1 = std::chrono::high_resolution_clock::now();
        if (useFloat)
            mul<<<ARRAY_SIZE/1024, 1024>>>((float*)d_b, (float*)d_c);
        else
            mul<<<ARRAY_SIZE/1024, 1024>>>((double*)d_b, (double*)d_c);
        check_cuda_error();
        hipDeviceSynchronize();
        check_cuda_error();
        t2 = std::chrono::high_resolution_clock::now();
        times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


        t1 = std::chrono::high_resolution_clock::now();
        if (useFloat)
            add<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_b, (float*)d_c);
        else
            add<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_b, (double*)d_c);
        check_cuda_error();
        hipDeviceSynchronize();
        check_cuda_error();
        t2 = std::chrono::high_resolution_clock::now();
        times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


        t1 = std::chrono::high_resolution_clock::now();
        if (useFloat)
            triad<<<ARRAY_SIZE/1024, 1024>>>((float*)d_a, (float*)d_b, (float*)d_c);
        else
            triad<<<ARRAY_SIZE/1024, 1024>>>((double*)d_a, (double*)d_b, (double*)d_c);
        check_cuda_error();
        hipDeviceSynchronize();
        check_cuda_error();
        t2 = std::chrono::high_resolution_clock::now();
        times.push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

        timings.push_back(times);

    }

    // Check solutions
    hipMemcpy(h_a, d_a, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
    check_cuda_error();
    hipMemcpy(h_b, d_b, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
    check_cuda_error();
    hipMemcpy(h_c, d_c, ARRAY_SIZE*DATATYPE_SIZE, hipMemcpyDeviceToHost);
    check_cuda_error();

    if (useFloat)
    {
        check_solution<float>(h_a, h_b, h_c);
    }
    else
    {
        check_solution<double>(h_a, h_b, h_c);
    }

    // Crunch results
    size_t sizes[4] = {
        2 * DATATYPE_SIZE * ARRAY_SIZE,
        2 * DATATYPE_SIZE * ARRAY_SIZE,
        3 * DATATYPE_SIZE * ARRAY_SIZE,
        3 * DATATYPE_SIZE * ARRAY_SIZE
    };
    double min[4] = {DBL_MAX, DBL_MAX, DBL_MAX, DBL_MAX};
    double max[4] = {0.0, 0.0, 0.0, 0.0};
    double avg[4] = {0.0, 0.0, 0.0, 0.0};

    // Ignore first result
    for (unsigned int i = 1; i < NTIMES; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            avg[j] += timings[i][j];
            min[j] = std::min(min[j], timings[i][j]);
            max[j] = std::max(max[j], timings[i][j]);
        }
    }

    for (int j = 0; j < 4; j++)
        avg[j] /= (double)(NTIMES-1);

    // Display results
    std::string labels[] = {"Copy", "Mul", "Add", "Triad"};
    std::cout
        << std::left << std::setw(12) << "Function"
        << std::left << std::setw(12) << "MBytes/sec"
        << std::left << std::setw(12) << "Min (sec)"
        << std::left << std::setw(12) << "Max"
        << std::left << std::setw(12) << "Average"
        << std::endl;

    for (int j = 0; j < 4; j++)
    {
        std::cout
            << std::left << std::setw(12) << labels[j]
            << std::left << std::setw(12) << std::setprecision(3) << 1.0E-06 * sizes[j]/min[j]
            << std::left << std::setw(12) << std::setprecision(5) << min[j]
            << std::left << std::setw(12) << std::setprecision(5) << max[j]
            << std::left << std::setw(12) << std::setprecision(5) << avg[j]
            << std::endl;
    }

    // Free host vectors
    free(h_a);
    free(h_b);
    free(h_c);

}

std::string getDeviceName(int device)
{
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    check_cuda_error();
    return std::string(prop.name);
}

void listDevices(void)
{
    // Get number of devices
    int count;
    hipGetDeviceCount(&count);
    check_cuda_error();

    // Print device names
    if (count == 0)
    {
        std::cout << "No devices found." << std::endl;
    }
    else
    {
        std::cout << std::endl;
        std::cout << "Devices:" << std::endl;
        for (int i = 0; i < count; i++)
        {
            std::cout << i << ": " << getDeviceName(i) << std::endl;
            check_cuda_error();
        }
        std::cout << std::endl;
    }
}

